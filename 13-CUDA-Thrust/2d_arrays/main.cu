#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>

#include <random>
#include <algorithm>

using namespace std;

const int ARRAY_SIZE = 100;
const int INNER_ARRAY_SIZE = 50;

void generate_random_array(int *array, size_t size);

int main() {
    int* two_dim_array1[ARRAY_SIZE];
    int** two_dim_array1_device = nullptr;
    for (auto &item: two_dim_array1) {
        int inner_array[INNER_ARRAY_SIZE];
        generate_random_array(inner_array, INNER_ARRAY_SIZE);
        item = inner_array;
    }
    hipMalloc((void**)&two_dim_array1_device, ARRAY_SIZE * sizeof(int*));
    for (int i = 0; i < ARRAY_SIZE; i++) {
        int* item_ptr = nullptr;
        hipMalloc((void**)&item_ptr, INNER_ARRAY_SIZE * sizeof(int));
        hipMemcpy(item_ptr, &two_dim_array1[i], INNER_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(&two_dim_array1_device[i], &item_ptr, sizeof(int*), hipMemcpyHostToDevice);
    }
    return 0;
}

void generate_random_array(int *array, size_t size) {
    random_device dev;
    default_random_engine engine(dev());
    uniform_int_distribution<int> uniform_dist(0, 100);
    generate(array, &array[size], [&] { return uniform_dist(engine); });
}