
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using namespace std;

void generate_data(int* data, size_t count);
__global__ void get_sum(const int *data, const size_t* count, int* sum);


/// The example randomly generates a huge array of ints and computes its sum on the GPU.

int main() {
    constexpr size_t ARRAY_SIZE = 10000000;
    // create an array that holds a large number of integers and fill it with data
    auto numbers = new int[ARRAY_SIZE];
    generate_data(numbers, ARRAY_SIZE);
    int initial_sum = 0; // this value is the initial value for sum: we start from zero and then add numbers to it

    // pointers that will point to device memory
    int *device_numbers, *device_sum;
    size_t *device_count;

    // allocate space for the array, array size and result. Before cudaMallocs the pointers are null pointers,
    // cudaMalloc changes their value to some GPU memory address, so we need to pass the address of the pointer, not the
    // pointer itself
    hipMalloc(&device_numbers, ARRAY_SIZE * sizeof(int));  // allocate memory chuck for the array
    hipMalloc(&device_count, sizeof(size_t));  // allocate memory for array size (one size_t)
    hipMalloc(&device_sum, sizeof(int));  // allocate memory for the result (one int)

    // copy data from RAM to VRAM
    hipMemcpy(device_numbers, numbers, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_count, &ARRAY_SIZE, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(device_sum, &initial_sum, sizeof(int), hipMemcpyHostToDevice);

    // get device properties, we will use it to launch the maximum available number of threads
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);

    // run get_sum on GPU
    get_sum<<<1, prop.maxThreadsPerBlock>>>(device_numbers, device_count, device_sum);

    int sum = 0;
    // get sum from VRAM
    hipMemcpy(&sum, device_sum, sizeof(int), hipMemcpyDeviceToHost);
    cout << sum << endl;

    // free all memory on the heap and VRAM
    delete [] numbers;
    hipFree(device_numbers);
    hipFree(device_sum);
    hipFree(device_count);
}

/// given an array and its size, fills it with random numbers from 0 to 1000
/// \param data - an array to fill
/// \param count - size of the array
void generate_data(int *data, const size_t count) {
    random_device rand;
    mt19937 rng(rand());
    uniform_int_distribution<int> uni(0, 1000);
    for (auto i = 0; i < count; i++) {
        data[i] = uni(rng);
    }
}

/// Computes the partial sum of the data array and adds it to the global sum
/// \param data - an array to get sum for
/// \param count - size of the array
/// \param sum - the global sum, this is the result of this function
__global__ void get_sum(const int *data, const size_t* count, int* sum) {
    // one thread processes one slice of data. Slice size is equal to total count divided by number of threads
    const auto slice_size = *count / blockDim.x;
    // compute start index
    const unsigned long start_index = slice_size * threadIdx.x;
    // compute end index. Last thread takes all remaining elements in case they are not split evenly between threads
    unsigned long end_index;
    if (threadIdx.x == blockDim.x - 1) {
        end_index = *count;
    } else {
        end_index = slice_size * (threadIdx.x + 1);
    }
    // compute sum of the chunk of the array
    auto local_sum = 0;
    for (auto i = start_index; i < end_index; i++) {
        local_sum += data[i];
    }
    // add the local sum to the global sum. This modifies shared (global) memory, so update it atomically
    atomicAdd(sum, local_sum);
}
