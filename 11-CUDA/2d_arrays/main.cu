#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <random>
#include <algorithm>

using namespace std;

constexpr size_t ARRAY_SIZE = 100;
constexpr size_t INNER_ARRAY_SIZE = 50;
constexpr size_t FULL_ARRAY_SIZE = ARRAY_SIZE * INNER_ARRAY_SIZE;

void generate_random_array(int *array, size_t size);
void print_matrix(const int* matrix);
__global__ void get_doubled_matrix(const int* original, int* result);

// a program that demonstrates how to use blocks and threads to multiply all elements in a matrix by 2.
int main() {
    const auto flat_matrix = new int[FULL_ARRAY_SIZE];  // matrix will be held in a flat array
    // pointers that will be assigned by cuda memory allocation
    int* flat_matrix_device;
    int* doubled_matrix;
    // result will be held in this array
    const auto doubled_matrix_host = new int[FULL_ARRAY_SIZE];
    // fill array with random values
    generate_random_array(flat_matrix, FULL_ARRAY_SIZE);
    // output what we generated
    print_matrix(flat_matrix);

    // make memory on GPU to fit our original data and write the address of that memory to flat_matrix_device pointer
    hipMalloc(&flat_matrix_device, FULL_ARRAY_SIZE * sizeof(int));
    // copy to flat_matrix_device from flat_matrix (from CPU to GPU memory)
    hipMemcpy(flat_matrix_device, flat_matrix, FULL_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
    // make memory on GPU for our result
    hipMalloc(&doubled_matrix, FULL_ARRAY_SIZE * sizeof(int));

    // launch ARRAY_SIZE blocks on GPU, each block containing INNER_ARRAY_SIZE of threads.
    get_doubled_matrix<<<ARRAY_SIZE, INNER_ARRAY_SIZE>>>(flat_matrix_device, doubled_matrix);
    // wait for CUDA operations to finish
    hipDeviceSynchronize();

    // retrieve our result from GPU
    hipMemcpy(doubled_matrix_host, doubled_matrix, FULL_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    // destroy allocated memory on GPU, we no longer need it
    hipFree(flat_matrix_device);
    hipFree(doubled_matrix);

    // some pretty output
    cout << setfill('-') << setw(INNER_ARRAY_SIZE * 4) << "" << endl << setfill(' ');
    print_matrix(doubled_matrix_host);

    // destroy allocated memory on CPU
    delete[] flat_matrix;
    delete[] doubled_matrix_host;

    return 0;
}

// a function that fills an array with random values
void generate_random_array(int *array, size_t size) {
    random_device dev;
    default_random_engine engine(dev());
    uniform_int_distribution<int> uniform_dist(0, 100);
    generate(array, &array[size], [&] { return uniform_dist(engine); });
}

// a function that prints a matrix nicely to console
void print_matrix(const int* matrix) {
    for (auto i = 0; i < ARRAY_SIZE; i++) {
        for (auto j = 0; j < INNER_ARRAY_SIZE; j++) {
            const auto index = INNER_ARRAY_SIZE * i + j;
            cout << setw(4) << matrix[index];
        }
        cout << endl;
    }
}

// Function that is run on GPU as many times as there elements in our matrix. One thread computes one element in the 
// result matrix.
__global__ void get_doubled_matrix(const int* original, int* result) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    result[index] = original[index] * 2;
}
