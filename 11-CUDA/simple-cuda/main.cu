#include "hip/hip_runtime.h"
#include <iostream>

using namespace std;

__global__ void run_on_gpu();
__device__ void execute(const char* name);

// a program that prints different strings in parallel from GPU
int main() {
    // start function run_on_gpu on one block on two threads concurrently
    run_on_gpu<<<1, 2>>>();
    // wait for device to finish
    hipDeviceSynchronize();
    cout << "Finished" << endl;
}

// a function that prints some text to console depending on its number. Global function means that it will be called
// from CPU code, but runs on GPU
__global__ void run_on_gpu() {
    const char* name;
    // thread id is between 0 and blockDim (2 in this case)
    if (threadIdx.x == 0) {
        name = "Thread 1";
    } else {
        name = "Thread 2";
    }
    execute(name);
}

// a function that prints some text to console on GPU
__device__ void execute(const char* name) {
    printf("%s: first\n", name);
    printf("%s: second\n", name);
    printf("%s: third\n", name);
}
