#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <algorithm>

using namespace std;

constexpr int ARRAY_SIZE = 100;

void generate_random_array(int* array, size_t size);
__global__ void add(const int* a, const int* b, int* c);

int main() {
    int first[ARRAY_SIZE], second[ARRAY_SIZE], sum[ARRAY_SIZE];
    generate_random_array(first, ARRAY_SIZE);
    generate_random_array(second, ARRAY_SIZE);
    int *device_first, *device_second, *device_sum;
    constexpr int size = ARRAY_SIZE * sizeof(int);
    hipMalloc(&device_first, size);
    hipMalloc(&device_second, size);
    hipMalloc(&device_sum, size);
    hipMemcpy(device_first, first, size, hipMemcpyHostToDevice);
    hipMemcpy(device_second, second, size, hipMemcpyHostToDevice);
    add<<<1, ARRAY_SIZE>>>(device_first, device_second, device_sum);
    hipDeviceSynchronize();
    hipMemcpy(sum, device_sum, size, hipMemcpyDeviceToHost);
    hipFree(device_first);
    hipFree(device_second);
    hipFree(device_sum);
    for_each(sum, &sum[ARRAY_SIZE], [](const int &n) { cout << n << endl;});
    return 0;
}

void generate_random_array(int *array, size_t size) {
    random_device dev;
    default_random_engine engine(dev());
    uniform_int_distribution<int> uniform_dist(0, 100);
    generate(array, &array[size], [&] { return uniform_dist(engine); });
}

__global__ void add(const int* a, const int* b, int* c) {
    if (const auto thread_id = threadIdx.x; thread_id < ARRAY_SIZE) {
        c[thread_id] = a[thread_id] + b[thread_id];
    }
}
