#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>


using namespace std;

__global__ void run_on_gpu();
__device__ void execute(const char* name);

int main() {
    run_on_gpu<<<1, 2>>>();
    hipDeviceSynchronize();
    cout << "Finished" << endl;
}

__global__ void run_on_gpu() {
    const char* name;
    if (threadIdx.x == 0) {
        name = "Thread 1";
    } else {
        name = "Thread 2";
    }
    execute(name);
}

__device__ void execute(const char* name) {
    printf("%s: first\n", name);
    printf("%s: second\n", name);
    printf("%s: third\n", name);
}
