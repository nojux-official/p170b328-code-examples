#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>

#include <random>
#include <algorithm>

using namespace std;

const int ARRAY_SIZE = 100;

void generate_random_array(int* array, size_t size);
__global__ void add(int* a, int* b, int* c);

int main() {
    int first[ARRAY_SIZE], second[ARRAY_SIZE], sum[ARRAY_SIZE];
    generate_random_array(first, ARRAY_SIZE);
    generate_random_array(second, ARRAY_SIZE);
    int *device_first, *device_second, *device_sum;
    int size = ARRAY_SIZE * sizeof(int);
    hipMalloc((void**)&device_first, size);
    hipMalloc((void**)&device_second, size);
    hipMalloc((void**)&device_sum, size);
    hipMemcpy(device_first, first, size, hipMemcpyHostToDevice);
    hipMemcpy(device_second, second, size, hipMemcpyHostToDevice);
    add<<<1, ARRAY_SIZE>>>(device_first, device_second, device_sum);
    hipDeviceSynchronize();
    hipMemcpy(sum, device_sum, size, hipMemcpyDeviceToHost);
    hipFree(device_first);
    hipFree(device_second);
    hipFree(device_sum);
    for_each(sum, &sum[ARRAY_SIZE], [](int &n) { cout << n << endl;});
    return 0;
}

void generate_random_array(int *array, size_t size) {
    random_device dev;
    default_random_engine engine(dev());
    uniform_int_distribution<int> uniform_dist(0, 100);
    generate(array, &array[size], [&] { return uniform_dist(engine); });
}

__global__ void add(int* a, int* b, int* c) {
    int thread_id = threadIdx.x;
    if (thread_id < ARRAY_SIZE) {
        c[thread_id] = a[thread_id] + b[thread_id];
    }
}
